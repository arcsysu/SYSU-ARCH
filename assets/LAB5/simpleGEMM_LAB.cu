#include "hip/hip_runtime.h"
/**
 * @author gtyinstinct
 * naive GEMM(int) implemention for demo
 * C = AB
 * A  (M X K) 
 * B  (K X N)
 * C  (M X N)
 **/

#include <cstdlib>
#include <cstdio>
#include <random>
#include <functional>

#define M 128               // matrix shape M in GEMM
#define N 64                // matrix shape N in GEMM
#define K 256               // matrix shape K in GEMM
#define RANGE 256           // data range is [-RANGE/2,RANGE/2)
#define TEST_TIMES 2        // iteration times
#define BLOCK_DIM_MIN 128   // minimum of BLOCK_DIM
#define BLOCK_DIM_MAX 1024  // maximum of BLOCK_DIM 
#define USE_TIMER 1         // 1 means use timer; 0 means without timer

// kernel function
// implement GEMM C=AB
__global__ void 
GEMM_gpu(int *A,int *B,int *C){
    // index in Grid scope
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // row index in C
    int row = idx / N;
    // column index in C
    int col = idx % N;
    // redundant thread do nothing
    if(row>=M)return;
    // init accumulation
    int sum = 0;
    // calculate C[row][col]
    for(int k=0;k<K;k++){
        sum += A[row*K + k] * B[k*N + col];
    }
    // write result to C
    C[row*N + col] = sum;
}

bool
check(int *C_cpu,int *C_gpu){
    for(int i=0;i<M;i++){
        for(int j=0;j<N;j++){
            if(C_cpu[i*N + j] != C_gpu[i*N + j]){
                printf("at (%d,%d) expect %d but got %d\n",i,j,C_cpu,C_gpu);
                return 0;
            }
        }
    }
    return 1;
}

void
GEMM_cpu(int *A,int *B,int *C){
    for(int i=0;i<M;i++){
        for(int j=0;j<N;j++){
            int sum = 0;
            for(int k=0;k<K;k++){
                sum += A[i*K + k] * B[k*N + j];
            }
            C[i*N +j] = sum;
        }
    }
}

void 
initVal(int *arr,size_t size){
    for(int i=0;i<size;i++){
        arr[i] = rand() % RANGE - RANGE/2; // [-RANGE/2,RANGE/2)
    }
}

void Timer(const char *tag, const std::function<void()> &kernel,
            int test_time=1) {
  for (int i = 0; i < test_time; ++i) {
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    hipEventRecord(beg);
    kernel();
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, beg, end);
    printf("[%s] iter %d: %f ms elapsed\n", tag, i,elapsed_time);
  }
}

int 
main(){
  #if USE_TIMER==1
    printf("Timer:ON\n");
  #else
    printf("Timer:OFF\n");
  #endif 
    printf("M:%d N:%d K:%d\n",M,N,K);

    // set magic number for seed
    srand(8);

    // malloc mem for A B C 
    // h:host pointer,d:device pointer
    int *A_h,*A_d,*B_h,*B_d,*C_h,*C_d;
    size_t A_size,B_size,C_size;
    // calculate size(byte) for A B C
    A_size = M*K*sizeof(int);
    B_size = K*N*sizeof(int);
    C_size = M*N*sizeof(int);
    // malloc on host CPU (row major)
    A_h = (int *)malloc(A_size);
    B_h = (int *)malloc(B_size);
    C_h = (int *)malloc(C_size);
    // malloc on device GPU (row major)
    hipMalloc(&A_d,A_size);
    hipMalloc(&B_d,B_size);
    hipMalloc(&C_d,C_size);

    // init val for A B at host memory
    initVal(A_h,A_size/sizeof(int));
    initVal(B_h,B_size/sizeof(int));

    // copy A B from host to device memory
    hipMemcpy(A_d,A_h,A_size,hipMemcpyHostToDevice);
    hipMemcpy(B_d,B_h,B_size,hipMemcpyHostToDevice);

    // launch kernel on GPU
    int GridDim,BlockDim;
    BlockDim = min(BLOCK_DIM_MAX,N); // let BlockDim <= BLOCK_DIM_MAX
    BlockDim = max(BLOCK_DIM_MIN,BlockDim); // let BlockDim >= BLOCK_DIM_MIN
    GridDim = M*N/BlockDim; // launch thread num M*N 
    GridDim = max(1,GridDim); // let GridDim >= 1
    printf("GridDim:%d BlockDim:%d\n",GridDim,BlockDim);
  #if USE_TIMER==1
    Timer("GEMM on GPU",[&]{GEMM_gpu<<<GridDim,BlockDim>>>(A_d,B_d,C_d);}
    ,TEST_TIMES);
  #else
    GEMM_gpu<<<GridDim,BlockDim>>>(A_d,B_d,C_d); 
  #endif
    

    // copy C from device to host memory
    hipMemcpy(C_h,C_d,C_size,hipMemcpyDeviceToHost);

    // Test GEMM on CPU
    int *C_cpu = (int *)malloc(C_size);
  #if USE_TIMER==1
    Timer("GEMM on CPU",[&]{GEMM_cpu(A_h,B_h,C_cpu);}
    ,TEST_TIMES);
  #else 
    GEMM_cpu(A_h,B_h,C_cpu);
  #endif

    // check result
    if(check(C_cpu,C_h)){
        printf("check pass!\n");
    }else{
        printf("check fail\n");
    }
}